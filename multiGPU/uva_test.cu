#include <stdio.h> 
#include <stdlib.h> 
#include <hip/hip_runtime.h>


__global__ void simpleKernel(float *dst, float *src1, float *src2)
{
  const int idx = blockIdx.x * blockDim.x + threadIdx.x;
  //float temp = src[idx];
  dst[idx] = src1[idx] + src2[idx];
}

int execute_uva(bool copy=false, bool print=false)
{
  float *src1,*src2, *dst;
  float *dsrc1, *dsrc2, *ddst, *dsrc2_1;
  size_t rsize = 256;
  size_t size = sizeof(float) * rsize * rsize;
  //cpu buffers
  src1 = (float *)malloc(size);
  src2 = (float *)malloc(size);
  dst = (float *)malloc(size);
  for (int i = 0; i < rsize * rsize; ++i) {
    src1[i] = (float)i;
    src2[i] = (float)(2 * i);
  }
  //gpu buffers
  hipSetDevice(0);
  hipDeviceEnablePeerAccess(1, 0);
  hipMalloc(&ddst, size);
  hipMalloc(&dsrc1, size);
  hipMemcpy(dsrc1, src1, size, hipMemcpyHostToDevice);
  // device setting here
  hipSetDevice(1);
  hipDeviceEnablePeerAccess(0, 0);
  hipMalloc(&dsrc2, size);
  hipMemcpy(dsrc2, src2, size, hipMemcpyHostToDevice);
  
  //Launch the kernel
  hipSetDevice(0);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start);
  if (copy) { 
    //Add all the overhead of copying to the times
    //including memory allocation
    hipMalloc(&dsrc2_1, size);
    hipMemcpy(dsrc2_1, dsrc2, size, hipMemcpyDefault);
    simpleKernel<<<rsize, rsize>>>(ddst, dsrc2_1, dsrc2);
  } else {
    simpleKernel<<<rsize, rsize>>>(ddst, dsrc1, dsrc2);
  }
  hipEventRecord(stop);
  hipEventSynchronize(stop);
  float time_ms = 0;
  hipEventElapsedTime(&time_ms, start, stop);
  hipMemcpy(dst, ddst, size, hipMemcpyDeviceToHost);
  if (print)
    for(int i = 0; i < rsize*rsize; ++i) {
      printf("%d: %f\n", i, dst[i]);
    }
  printf("Last item: %f\n", dst[rsize*rsize-1]);
  printf("Elapsed time: %f\n", time_ms); 
  // clean gpu buffers
  hipFree(ddst);
  hipFree(dsrc1);
  if (copy) hipFree(dsrc2_1);
  // Just in case
  hipSetDevice(1);
  hipFree(dsrc2);
  // clean cpu buffers
  free(src1);
  free(src2);
  free(dst);
  return 0;
}


int main()
{
  int canAccess10, canAccess01;
  hipDeviceCanAccessPeer(&canAccess10, 1, 0);
  printf("Access status: %d\n", canAccess10);
  hipDeviceCanAccessPeer(&canAccess01, 0, 1);
  printf("Access status: %d\n", canAccess01);
  if (canAccess10 && canAccess01) {
    execute_uva(true);
  }
  return 0;
}
